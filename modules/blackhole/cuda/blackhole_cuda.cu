#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <vector>
#include ""
#include "blackhole_cuda.h"

__constant__ float PI = 3.1415926535897932384626433832795f;

__device__ void geodesic(float u, float dudphi, float& out_du_dphi, float& out_d2u_dphi2, float rs) {
    out_du_dphi = dudphi;
    out_d2u_dphi2 = -u * (1.f - 3.f / 2.f * rs * u);
}

__device__ void rk4_step(float& u, float& dudphi, float& phi, float h, float rs) {
    float k1_u, k1_dudphi, k2_u, k2_dudphi, k3_u, k3_dudphi, k4_u, k4_dudphi;

    geodesic(u, dudphi, k1_u, k1_dudphi, rs);
    geodesic(u + 0.5f * k1_u * h, dudphi + 0.5f * k1_dudphi * h, k2_u, k2_dudphi, rs);
    geodesic(u + 0.5f * k2_u * h, dudphi + 0.5f * k2_dudphi * h, k3_u, k3_dudphi, rs);
    geodesic(u + k3_u * h, dudphi + k3_dudphi * h, k4_u, k4_dudphi, rs);

    phi += h;
    u = u + (k1_u + 2.f * k2_u + 2.f * k3_u + k4_u) * h / 6.f;
    dudphi = dudphi + (k1_dudphi + 2.f * k2_dudphi + 2.f * k3_dudphi + k4_dudphi) * h / 6.f;
}

__global__ void solveGeodesicKernel(float rs, float u_0, float* dudphi_0_values, float h, float* envmap_r_values, size_t num_envmaps, size_t num_rays, size_t num_steps, float* angles_out) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_rays) return;

    size_t const outNodeSize = num_envmaps + 1;

    float u = u_0;
    float dudphi = dudphi_0_values[idx];
    float phi = 0.0f;

    // Perform the first RK4 step
    rk4_step(u, dudphi, phi, h, rs);

    float r = 1.0f / u;
    float const r_0 = 1.0f / u_0;
    float a = r * sin(phi);
    float b = r * cos(phi) - r_0;

    // Store starting angle (angle is local to camera)
    angles_out[idx * outNodeSize] = atan2(a, b);

    auto out_of_bounds = [&u, &envmap_r_values](size_t index) -> bool {
        return (1.0f / u > envmap_r_values[index]);
        };

    auto inside_singularity = [&u, &rs]() -> bool {
        return (1.0f / u <= rs);
        };

    size_t env_index = 0;
    for (size_t step = 1; step < num_steps && !out_of_bounds(num_envmaps-1) && !inside_singularity(); step++) {
        rk4_step(u, dudphi, phi, h, rs);
        if (out_of_bounds(env_index)) {
            angles_out[idx * outNodeSize + 1 + env_index++] = phi;
        }
    }
    angles_out[idx * outNodeSize + num_envmaps] = phi;
    for (size_t i = 0; i < num_envmaps; ++i) {
        angles_out[idx * outNodeSize + 1 + i] = !inside_singularity() ? angles_out[idx * outNodeSize + 1 + i] : nan("");
    }
}

void generate_du(float* d_du_0_values, float min, float max, size_t count) {
    for (size_t i = 0; i < count; ++i) {
        float t = 2.0f * (i / static_cast<float>(count - 1)) - 1.0f;
        float smooth_t = (t < 0.0f ? -1.0f : 1.0f) * (powf(fabsf(t), 3.0f));
        d_du_0_values[i] = min + (smooth_t + 1.0f) * (max - min) / 2.0f;
    }  
}

void schwarzchild(
    float rs, std::vector<float> envmap_r_values, size_t num_rays, size_t num_steps, float u_0, float h, std::vector<float>& angle_out) {

    float* d_dudphi_0_values;
    float* d_angle_values;
    float* d_envmap_r_values;

    size_t const outValuesPerRay = (envmap_r_values.size() + 1);
    // Allocate device memory
    hipMalloc(&d_dudphi_0_values, num_rays * sizeof(float));
    hipMalloc(&d_angle_values, num_rays * outValuesPerRay * sizeof(float));
    hipMalloc(&d_envmap_r_values, envmap_r_values.size() * sizeof(float));

    // Copy initial velocity values to device
    std::vector<float> dudphi_0_values(num_rays, 0.f);
    generate_du(dudphi_0_values.data(), 50000, -50000, num_rays);
    hipMemcpy(d_dudphi_0_values, dudphi_0_values.data(), num_rays * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_envmap_r_values, envmap_r_values.data(), envmap_r_values.size() * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 256;
    int numBlocks = (num_rays + threadsPerBlock - 1) / threadsPerBlock;
    solveGeodesicKernel<<<numBlocks, threadsPerBlock>>>(rs, u_0, d_dudphi_0_values, h, d_envmap_r_values, envmap_r_values.size(), num_rays, num_steps, d_angle_values);
    angle_out.resize(num_rays * outValuesPerRay, std::numeric_limits<double>::quiet_NaN());
    hipMemcpy(angle_out.data(), d_angle_values, num_rays * outValuesPerRay * sizeof(float), hipMemcpyDeviceToHost);

    // Add handeling of special case straight backwards
    //for (int i = 0; i < envmap_r_values.size(); ++i) {
    //    angle_out[(num_rays - 1) * outValusPerRay] = 0.0f;
    //    angle_out[(num_rays - 1) * outValusPerRay + 1 + i] = 0.0f;
    //}

    hipFree(d_dudphi_0_values);
    hipFree(d_envmap_r_values);
    hipFree(d_angle_values);
}
