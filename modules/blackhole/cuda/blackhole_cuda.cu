#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <vector>
#include ""

__constant__ double rs = 1;

__device__ void geodesic(double u, double du, double& out_u, double& out_du) {
    double temp_u = u;
    out_u = du;
    out_du = -temp_u * (1 - (3 / 2) * rs * temp_u);
}

__device__ void rk4_step(double& u, double& du, double& phi, double h) {
    double k1_u, k1_du, k2_u, k2_du, k3_u, k3_du, k4_u, k4_du;

    geodesic(u, du, k1_u, k1_du);
    geodesic(u + 0.5 * k1_u * h, du + 0.5 * k1_du * h, k2_u, k2_du);
    geodesic(u + 0.5 * k2_u * h, du + 0.5 * k2_du * h, k3_u, k3_du);
    geodesic(u + k3_u * h, du + k3_du * h, k4_u, k4_du);

    phi += h;
    u = u + (k1_u + 2 * k2_u + 2 * k3_u + k4_u) * h / 6;
    du = du + (k1_du + 2 * k2_du + 2 * k3_du + k4_du) * h / 6;
}

__global__ void solve_geodesic_kernel(double u_0, double* du_0_values, double h, int num_paths, int num_steps, double* u_values, double* du_values, double* phi_values) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_paths) return;

    double u = u_0;
    double du = du_0_values[idx];
    double phi = 0.0;

    for (int step = 0; step < num_steps; step++) {
        u_values[idx * num_steps + step] = u;
        du_values[idx * num_steps + step] = du;
        phi_values[idx * num_steps + step] = phi;

        rk4_step(u, du, phi, h);
    }
}

extern "C" {
    __declspec(dllexport) void cuda_test(
        int num_paths, int num_steps, double u_0,
        double* du_0_values, double h, double* u_out, double* phi_out) {

        // Allocate device memory
        double* d_du_0_values;
        double* d_u_values;
        double* d_du_values;
        double* d_phi_values;

        hipMalloc(&d_du_0_values, num_paths * sizeof(double));
        hipMalloc(&d_u_values, num_paths * num_steps * sizeof(double));
        hipMalloc(&d_du_values, num_paths * num_steps * sizeof(double));
        hipMalloc(&d_phi_values, num_paths * num_steps * sizeof(double));


        // Copy initial velocity values to device
        hipMemcpy(d_du_0_values, du_0_values, num_paths * sizeof(double), hipMemcpyHostToDevice);

        // Launch kernel
        int threadsPerBlock = 256;
        int numBlocks = (num_paths + threadsPerBlock - 1) / threadsPerBlock;
        solve_geodesic_kernel << <numBlocks, threadsPerBlock >> > (u_0, d_du_0_values, h, num_paths, num_steps, d_u_values, d_du_values, d_phi_values);

        // Copy results back to host
        hipMemcpy(u_out, d_u_values, num_paths * num_steps * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(phi_out, d_phi_values, num_paths * num_steps * sizeof(double), hipMemcpyDeviceToHost);

        hipFree(d_du_0_values);
        hipFree(d_u_values);
        hipFree(d_du_values);
        hipFree(d_phi_values);
    }
}
