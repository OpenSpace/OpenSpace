#include "hip/hip_runtime.h"
#include "blackhole_cuda.h"
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void test_kernal() {
    printf("HelLo form GPU!!!");
}

void openspace::cuda_test()
{
    test_kernal<<<1, 1>>>();
}
