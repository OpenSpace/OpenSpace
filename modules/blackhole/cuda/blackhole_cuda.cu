#include <hip/hip_runtime.h>
#include <vector>


__constant__ double PI = 3.1415926535897932384626433832795;

__device__ void geodesic(double u, double dudphi, double& out_du_dphi, double& out_d2u_dphi2, double rs) {
    out_du_dphi = dudphi;
    out_d2u_dphi2 = -u * (1 - 3. / 2. * rs * u);
}

__device__ void rk4_step(double& u, double& dudphi, double& phi, double h, double rs) {
    double k1_u, k1_dudphi, k2_u, k2_dudphi, k3_u, k3_dudphi, k4_u, k4_dudphi;

    geodesic(u, dudphi, k1_u, k1_dudphi, rs);
    geodesic(u + 0.5 * k1_u * h, dudphi + 0.5 * k1_dudphi * h, k2_u, k2_dudphi, rs);
    geodesic(u + 0.5 * k2_u * h, dudphi + 0.5 * k2_dudphi * h, k3_u, k3_dudphi, rs);
    geodesic(u + k3_u * h, dudphi + k3_dudphi * h, k4_u, k4_dudphi, rs);

    phi += h;
    u = u + (k1_u + 2 * k2_u + 2 * k3_u + k4_u) * h / 6;
    dudphi = dudphi + (k1_dudphi + 2 * k2_dudphi + 2 * k3_dudphi + k4_dudphi) * h / 6;
}

__global__ void solveGeodesicKernel(double rs, double envmap_r, double u_0, double* dudphi_0_values, double h, int num_paths, int num_steps, double* angles_out) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_paths) return;

    double u = u_0;
    double dudphi = dudphi_0_values[idx];
    double phi = 0.0;

    // Perform the first RK4 step
    rk4_step(u, dudphi, phi, h, rs);

    double r = 1.0 / u;
    double r_0 = 1.0 / u_0;
    double a = r * sin(phi);
    double b = r_0 - r * cos(phi);

    // Store starting angle (angle is local to camera)
    angles_out[idx * 2] = atan2(a, b);

    auto out_of_bounds = [&u, &envmap_r]() -> bool {
        return (1.0 / u > envmap_r);
        };

    auto inside_singularity = [&u, &rs]() -> bool {
        return (1.0 / u <= rs);
        };

    for (int step = 1; step < num_steps && !out_of_bounds() && !inside_singularity(); step++) {
        rk4_step(u, dudphi, phi, h, rs);
    }

    angles_out[idx * 2 + 1] = !inside_singularity() ? phi : nan("");
}

void generate_du(double* d_du_0_values, double min, double max, int count) {
    double step = (max - min) / (count - 1);
    for (int i = 0; i < count; ++i) {
        d_du_0_values[i] = min + step * i;
    }
}

void schwarzchild(
    double rs, double envmap_r, int num_paths, int num_steps, double u_0, double h, double* angle_out) {

    double* d_dudphi_0_values;
    double* d_angle_values;

    // Allocate device memory
    hipMalloc(&d_dudphi_0_values, num_paths * sizeof(double));
    hipMalloc(&d_angle_values, num_paths * 2 * sizeof(double));

    // Copy initial velocity values to device
    std::vector<double> dudphi_0_values(num_paths, 0);
    generate_du(dudphi_0_values.data(), 1, -1, num_paths);
    hipMemcpy(d_dudphi_0_values, dudphi_0_values.data(), num_paths * sizeof(double), hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 256;
    int numBlocks = (num_paths + threadsPerBlock - 1) / threadsPerBlock;
    solveGeodesicKernel<<<numBlocks, threadsPerBlock>>>(rs, envmap_r, u_0, d_dudphi_0_values, h, num_paths, num_steps, d_angle_values);

    hipMemcpy(angle_out, d_angle_values, num_paths * 2 * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_dudphi_0_values);
    hipFree(d_angle_values);
}
