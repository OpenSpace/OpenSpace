#include <hip/hip_runtime.h>
#include <vector>


__constant__ float PI = 3.1415926535897932384626433832795f;

__device__ void geodesic(float u, float dudphi, float& out_du_dphi, float& out_d2u_dphi2, float rs) {
    out_du_dphi = dudphi;
    out_d2u_dphi2 = -u * (1.f - 3.f / 2.f * rs * u);
}

__device__ void rk4_step(float& u, float& dudphi, float& phi, float h, float rs) {
    float k1_u, k1_dudphi, k2_u, k2_dudphi, k3_u, k3_dudphi, k4_u, k4_dudphi;

    geodesic(u, dudphi, k1_u, k1_dudphi, rs);
    geodesic(u + 0.5f * k1_u * h, dudphi + 0.5f * k1_dudphi * h, k2_u, k2_dudphi, rs);
    geodesic(u + 0.5f * k2_u * h, dudphi + 0.5f * k2_dudphi * h, k3_u, k3_dudphi, rs);
    geodesic(u + k3_u * h, dudphi + k3_dudphi * h, k4_u, k4_dudphi, rs);

    phi += h;
    u = u + (k1_u + 2.f * k2_u + 2.f * k3_u + k4_u) * h / 6.f;
    dudphi = dudphi + (k1_dudphi + 2.f * k2_dudphi + 2.f * k3_dudphi + k4_dudphi) * h / 6.f;
}

__global__ void solveGeodesicKernel(float rs, float envmap_r, float u_0, float* dudphi_0_values, float h, int num_paths, int num_steps, float* angles_out) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_paths) return;

    float u = u_0;
    float dudphi = dudphi_0_values[idx];
    float phi = 0.0f;

    // Perform the first RK4 step
    rk4_step(u, dudphi, phi, h, rs);

    float r = 1.0f / u;
    float r_0 = 1.0f / u_0;
    float a = r * sin(phi);
    float b = r * cos(phi) - r_0;

    // Store starting angle (angle is local to camera)
    angles_out[idx * 2] = atan2(a, b);

    auto out_of_bounds = [&u, &envmap_r]() -> bool {
        return (1.0f / u > envmap_r);
        };

    auto inside_singularity = [&u, &rs]() -> bool {
        return (1.0f / u <= rs);
        };

    for (int step = 1; step < num_steps && !out_of_bounds() && !inside_singularity(); step++) {
        rk4_step(u, dudphi, phi, h, rs);
    }

    angles_out[idx * 2 + 1] = !inside_singularity() ? phi : nan("");
}

void generate_du(float* d_du_0_values, float min, float max, int count) {
    float step = (max - min) / (count - 1);
    for (int i = 0; i < count; ++i) {
        d_du_0_values[i] = min + step * i;
    }
}

void schwarzchild(
    float rs, float envmap_r, int num_paths, int num_steps, float u_0, float h, float* angle_out) {

    float* d_dudphi_0_values;
    float* d_angle_values;

    // Allocate device memory
    hipMalloc(&d_dudphi_0_values, num_paths * sizeof(float));
    hipMalloc(&d_angle_values, num_paths * 2 * sizeof(float));

    // Copy initial velocity values to device
    std::vector<float> dudphi_0_values(num_paths, 0.f);
    generate_du(dudphi_0_values.data(), 1, -1, num_paths);
    hipMemcpy(d_dudphi_0_values, dudphi_0_values.data(), num_paths * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 256;
    int numBlocks = (num_paths + threadsPerBlock - 1) / threadsPerBlock;
    solveGeodesicKernel<<<numBlocks, threadsPerBlock>>>(rs, envmap_r, u_0, d_dudphi_0_values, h, num_paths, num_steps, d_angle_values);

    hipMemcpy(angle_out, d_angle_values, num_paths * 2 * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_dudphi_0_values);
    hipFree(d_angle_values);
}
