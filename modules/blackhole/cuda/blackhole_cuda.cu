#include <hip/hip_runtime.h>
#include <vector>


__constant__ float PI = 3.1415926535897932384626433832795f;

__device__ void geodesic(float u, float dudphi, float& out_du_dphi, float& out_d2u_dphi2, float rs) {
    out_du_dphi = dudphi;
    out_d2u_dphi2 = -u * (1.f - 3.f / 2.f * rs * u);
}

__device__ void rk4_step(float& u, float& dudphi, float& phi, float h, float rs) {
    float k1_u, k1_dudphi, k2_u, k2_dudphi, k3_u, k3_dudphi, k4_u, k4_dudphi;

    geodesic(u, dudphi, k1_u, k1_dudphi, rs);
    geodesic(u + 0.5f * k1_u * h, dudphi + 0.5f * k1_dudphi * h, k2_u, k2_dudphi, rs);
    geodesic(u + 0.5f * k2_u * h, dudphi + 0.5f * k2_dudphi * h, k3_u, k3_dudphi, rs);
    geodesic(u + k3_u * h, dudphi + k3_dudphi * h, k4_u, k4_dudphi, rs);

    phi += h;
    u = u + (k1_u + 2.f * k2_u + 2.f * k3_u + k4_u) * h / 6.f;
    dudphi = dudphi + (k1_dudphi + 2.f * k2_dudphi + 2.f * k3_dudphi + k4_dudphi) * h / 6.f;
}

__global__ void solveGeodesicKernel(float rs, float envmap_r, float u_0, float* dudphi_0_values, float h, size_t num_rays, size_t num_steps, float* angles_out) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_rays) return;

    float u = u_0;
    float dudphi = dudphi_0_values[idx];
    float phi = 0.0f;

    // Perform the first RK4 step
    rk4_step(u, dudphi, phi, h, rs);

    float r = 1.0f / u;
    float const r_0 = 1.0f / u_0;
    float a = r * sin(phi);
    float b = r * cos(phi) - r_0;

    // Store starting angle (angle is local to camera)
    angles_out[idx * 2] = atan2(a, b);

    auto out_of_bounds = [&u, &envmap_r]() -> bool {
        return (1.0f / u > envmap_r);
        };

    auto inside_singularity = [&u, &rs]() -> bool {
        return (1.0f / u <= rs);
        };

    for (size_t step = 1; step < num_steps && !out_of_bounds() && !inside_singularity(); step++) {
        rk4_step(u, dudphi, phi, h, rs);
    }

    angles_out[idx * 2 + 1] = !inside_singularity() ? phi : nan("");
}

void generate_du(float* d_du_0_values, float min, float max, size_t count) {
    for (size_t i = 0; i < count; ++i) {
        float t = 2.0f * (i / static_cast<float>(count - 1)) - 1.0f;
        float smooth_t = (t < 0.0f ? -1.0f : 1.0f) * (powf(fabsf(t), 3.0f));
        d_du_0_values[i] = min + (smooth_t + 1.0f) * (max - min) / 2.0f;
    }  
}

void schwarzchild(
    float rs, float envmap_r, size_t num_rays, size_t num_steps, float u_0, float h, float* angle_out) {

    float* d_dudphi_0_values;
    float* d_angle_values;

    // Allocate device memory
    hipMalloc(&d_dudphi_0_values, num_rays * sizeof(float));
    hipMalloc(&d_angle_values, num_rays * 2 * sizeof(float));

    // Copy initial velocity values to device
    std::vector<float> dudphi_0_values(num_rays, 0.f);
    generate_du(dudphi_0_values.data(), 50000, -50000, num_rays);
    hipMemcpy(d_dudphi_0_values, dudphi_0_values.data(), num_rays * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 256;
    int numBlocks = (num_rays + threadsPerBlock - 1) / threadsPerBlock;
    solveGeodesicKernel<<<numBlocks, threadsPerBlock>>>(rs, envmap_r, u_0, d_dudphi_0_values, h, num_rays, num_steps, d_angle_values);

    hipMemcpy(angle_out, d_angle_values, num_rays * 2 * sizeof(float), hipMemcpyDeviceToHost);

    // Add handeling of special case straight backwards
    angle_out[(num_rays - 1) * 2] = 0.0f;
    angle_out[(num_rays - 1) * 2 + 1] = 0.0f;

    hipFree(d_dudphi_0_values);
    hipFree(d_angle_values);
}
