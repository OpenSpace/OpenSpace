#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <vector>
#include <stdexcept>
#include ""
#include "schwarzschild.h"

#define HORIZION nanf("")

constexpr float PI = 3.1415926535897932384626433832795f;

__device__ float zamoconv(float r0, float psi) {
    // Compute the Schwarzschild lapse factor at r0:
    // For rs = 1, we have: N(r0) = sqrt(1 - 1/r0)
    float N_r0 = sqrtf(1.f - 1.f / r0);
    return -( N_r0 * cosf(psi) ) / ( r0 * sinf(psi) );
}

__device__ float compute_shadow_angle(float r_0) {
    // Critical impact parameter for a Schwarzschild black hole in geometric units.
    float b_crit = 3.0f * sqrtf(3.0f) / 2.0f;  // ~2.598
    // Compute the sine of the shadow angle:
    float ratio = b_crit * sqrtf(1.0f - 1.0f / r_0) / r_0;
    // Clamp to [0,1] to avoid domain errors
    ratio = fminf(fmaxf(ratio, 0.0f), 1.0f);
    return asinf(ratio);
}

__device__ void geodesic(float u, float dudphi, float& out_du_dphi, float& out_d2u_dphi2) {
    out_du_dphi = dudphi;
    out_d2u_dphi2 = -u * (1.f - 3.f / 2.f * u);
}

__device__ void rk4_step(float& u, float& dudphi, float& phi, float h) {
    float k1_u, k1_dudphi, k2_u, k2_dudphi, k3_u, k3_dudphi, k4_u, k4_dudphi;

    geodesic(u, dudphi, k1_u, k1_dudphi);
    geodesic(u + 0.5f * k1_u * h, dudphi + 0.5f * k1_dudphi * h, k2_u, k2_dudphi);
    geodesic(u + 0.5f * k2_u * h, dudphi + 0.5f * k2_dudphi * h, k3_u, k3_dudphi);
    geodesic(u + k3_u * h, dudphi + k3_dudphi * h, k4_u, k4_dudphi);

    phi += h;
    u = u + (k1_u + 2.f * k2_u + 2.f * k3_u + k4_u) * h / 6.f;
    dudphi = dudphi + (k1_dudphi + 2.f * k2_dudphi + 2.f * k3_dudphi + k4_dudphi) * h / 6.f;
}

__global__ void solveGeodesicKernel(float u_0, float h, float* envmap_r_values, size_t num_envmaps,
    size_t num_rays, size_t num_steps, float* angles_out) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_rays) return;

    size_t const outNodeSize = num_envmaps + 1;

    float r_0 = 1.0f / u_0;

    float theta_shadow = compute_shadow_angle(r_0);  // edge of the blackhole shadow

    const float delta = 0.01f;

    // Set the lower bound slightly inside the shadow:
    float lower_bound = theta_shadow - delta;

    float s = idx / static_cast<float>(num_rays - 1);
    float alpha = (PI - lower_bound) * (1.0f - s) + delta * s;

    angles_out[idx * outNodeSize] = alpha;

    // Compute initial derivative for this ray using the selected alpha.
    float dudphi = zamoconv(r_0, alpha); //compute_initial_dudphi(r_0, alpha);

    float u = u_0;
    float phi = 0.0f;

    auto out_of_bounds = [&u, &envmap_r_values](unsigned int index) -> bool {
        return (1.0f > envmap_r_values[index] * u);
        };

    auto inside_singularity = [&u]() -> bool {
        return (1.0f < u);
        };

    float* entry = &angles_out[idx * outNodeSize];

    unsigned int idx_entry = 0;
    entry[idx_entry++] = alpha;
    for (size_t step = 0; step < num_steps; step++) {
        rk4_step(u, dudphi, phi, h);

        bool in_singularity = inside_singularity();
        bool is_out_of_bounds = out_of_bounds(idx_entry - 1);
        if (in_singularity | is_out_of_bounds) {
            if (in_singularity) {
                while (idx_entry < outNodeSize) {
                    entry[idx_entry++] = HORIZION;
                }
                break;
            }
            if (is_out_of_bounds) {
                entry[idx_entry++] = phi;
                if (idx_entry > num_envmaps) {
                    break;
                }
            }
        }
    }
}

void schwarzschild(std::vector<float> const& envmap_r_values, size_t const num_rays, size_t const num_steps, float const r_0, float const h, std::vector<float>& angle_out) {
    float* d_angle_values;
    float* d_envmap_r_values;

    float u_0 = 1.0f / r_0;
    size_t const outValuesPerRay = envmap_r_values.size() + 1;

    hipMalloc(&d_angle_values, num_rays * outValuesPerRay * sizeof(float));
    hipMalloc(&d_envmap_r_values, envmap_r_values.size() * sizeof(float));
    hipMemcpy(d_envmap_r_values, envmap_r_values.data(), envmap_r_values.size() * sizeof(float), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int numBlocks = (num_rays + threadsPerBlock - 1) / threadsPerBlock;

    // Solve geodesics on GPU
    solveGeodesicKernel << <numBlocks, threadsPerBlock >> > (u_0, h, d_envmap_r_values, envmap_r_values.size(), num_rays, num_steps, d_angle_values);
    hipDeviceSynchronize();

    angle_out.resize(num_rays * outValuesPerRay);
    hipMemcpy(angle_out.data(), d_angle_values, num_rays * outValuesPerRay * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_envmap_r_values);
    hipFree(d_angle_values);
}
