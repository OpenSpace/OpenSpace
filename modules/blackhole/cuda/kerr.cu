#include "hip/hip_runtime.h"
#include "kerr.h"
#include <hip/hip_runtime.h>
#include ""
#include "vector_functions.h"
#include <cmath>
#include <cstdio>
#include <vector>

#ifndef M_PI
#define M_PI 3.14159265358979323846f
#endif

#ifndef M_C
#define M_C 299792458.0f
#endif

// ---------------------------------------------------------------------
// Device constants (set at compile time; you may also update via hipMemcpyToSymbol)
__constant__ float c_a = 0.99f;
__constant__ float c_rs = 1;
__constant__ unsigned int c_num_steps = 15000;
__constant__ unsigned int c_layers = 1;
__constant__ float c_M = 1.0f;     // Mass parameter
__constant__ float c_epsilon = 1e-10;   // Numerical tolerance
__constant__ float3 worldUp = { 0.0f, 0.0f, 1.0f };



// Additional simulation parameters
__constant__ float c_h = 0.01f;            // Integration step size

// ---------------------------------------------------------------------
// Coordinate convertion functions

// helper math (as before)
__device__ float3 crossf3(const float3& a, const float3& b) {
    return make_float3(
        a.y * b.z - a.z * b.y,
        a.z * b.x - a.x * b.z,
        a.x * b.y - a.y * b.x
    );
}

__device__ float3 normalizef3(const float3& v) {
    float len2 = v.x * v.x + v.y * v.y + v.z * v.z;
    float invLen = 1.0f / sqrtf(len2);
    return make_float3(v.x * invLen,
        v.y * invLen,
        v.z * invLen);
}


__device__ inline float3 operator*(const float3& v, float s) {
    return make_float3(v.x * s, v.y * s, v.z * s);
}

__device__ inline float3 operator*(float s, const float3& v) {
    return make_float3(v.x * s, v.y * s, v.z * s);
}

__device__ inline float3 operator+(const float3& a, const float3& b) {
    return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__device__ float3 spherical_to_cartesian(float r, float theta, float phi) {
    return make_float3(
        r * sinf(theta) * cosf(phi),
        r * sinf(theta) * sinf(phi),
        r * cosf(theta)
    );
}

__device__ void cartesian_to_boyer_lindquist(float x, float x_vel,
    float y, float y_vel,
    float z, float z_vel,
    float A, float* out) {
    double r2 = x * x + y * y + z * z;
    double A2 = A * A;
    double root = sqrt(A2 * (A2 - 2.0 * (x * x + y * y) + 2.0 * z * z) + r2 * r2);
    double radius = sqrt((-A2 + r2 + root) * 0.5);

    float azimuthal_angle = atan2f(y, x);
    float polar_angle = acosf(z / radius);

    double denom = 2.0 * radius * radius + A2 - r2;
    double radius_velocity = (radius * (x * x_vel + y * y_vel + z * z_vel)) / denom +
        A2 * z * z_vel / (radius * denom);

    float polar_denom = radius * sqrtf(radius * radius - z * z);
    float polar_velocity = (z * radius_velocity - z_vel * radius) / polar_denom;

    float azimuthal_velocity = (y_vel * x - x_vel * y) / (x * x + y * y);

    out[0] = radius;
    out[1] = radius_velocity;
    out[2] = polar_angle;
    out[3] = polar_velocity;
    out[4] = azimuthal_angle;
    out[5] = azimuthal_velocity;
}

// ---------------------------------------------------------------------
// Kerr metric helper functions

__device__ float sigma(float r, float theta) {
    float cos_theta = cos(theta);
    return r * r + c_a * c_a * cos_theta * cos_theta;
}

__device__ float delta_r(float r) {
    return r * r + c_a * c_a - 2.0f * c_M * r;
}

__device__ float ddelta_r(float r) {
    return 2.0f * (r - c_M);
}

// ---------------------------------------------------------------------
// Functions W_r, W_theta and their derivatives

__device__ float W_r(float r, float E, float L) {
    return E * (r * r + c_a * c_a) - c_a * L;
}

__device__ float dWsquare_r(float r, float E, float L) {
    float W = W_r(r, E, L);
    float dW_dr = 2.0f * E * r;
    return 2.0f * W * dW_dr;
}

__device__ float W_theta(float theta, float E, float L) {
    float sin_theta = sin(theta);
    sin_theta = fmax(sin_theta, c_epsilon);
    return c_a * E * sin_theta - L / sin_theta;
}

__device__ float dWsquare_theta(float theta, float E, float L) {
    float sin_theta = sin(theta);
    float cos_theta = cos(theta);
    sin_theta = fmax(sin_theta, c_epsilon);
    float dW_dtheta = cos_theta * (c_a * E + L / (sin_theta * sin_theta));
    return 2.0f * W_theta(theta, E, L) * dW_dtheta;
}

// ---------------------------------------------------------------------
// Definitions of the conserved quantities and derived functions

__device__ float E_func(float r, float theta, float dr, float dtheta, float dphi) {
    float sin_theta = sin(theta);
    sin_theta = fmax(sin_theta, c_epsilon);
    float delta = delta_r(r);
    float term = ((c_a * c_a * sin_theta * sin_theta - delta) * (-dr * dr / delta - dtheta * dtheta)
        + (dphi * sin_theta) * (dphi * sin_theta) * delta);
    return sqrt(term);
}

__device__ float L_func(float r, float theta, float dphi, float E) {
    float sin_theta = sin(theta);
    sin_theta = fmax(sin_theta, c_epsilon);
    float delta = delta_r(r);
    float sigma_val = sigma(r, theta);
    float num = c_a * E * delta + (sigma_val * delta * dphi - c_a * E * (r * r + c_a * c_a));
    float denom = delta - c_a * c_a * sin_theta * sin_theta;
    return sin_theta * sin_theta * num / denom;
}

__device__ float k_func(float r, float theta, float dr, float E, float L) {
    float sigma_val = sigma(r, theta);
    float delta = delta_r(r);
    float W = W_r(r, E, L);
    return (W * W - sigma_val * sigma_val * dr * dr) / delta;
}

// ---------------------------------------------------------------------
// Geodesic equations: state vector y = [r, theta, phi, p_r, p_theta]
__device__ float dr_func(float r, float theta, float p_r) {
    return delta_r(r) * p_r / sigma(r, theta);
}

__device__ float dtheta_func(float r, float theta, float p_theta) {
    return p_theta / sigma(r, theta);
}

__device__ float dphi_func(float r, float theta, float E, float L) {
    float sig = sigma(r, theta);
    float delta = delta_r(r);
    float sin_theta = sin(theta);
    sin_theta = fmax(sin_theta, c_epsilon);
    return (c_a * W_r(r, E, L) / delta - W_theta(theta, E, L) / sin_theta) / sig;
}

__device__ float dp_r(float r, float theta, float p_r, float E, float L, float k_val) {
    float sig = sigma(r, theta);
    float delta = delta_r(r);
    float d_delta = ddelta_r(r);
    float dW2 = dWsquare_r(r, E, L);
    float num = dW2 - d_delta * k_val;
    return (num / (2.0f * delta) - d_delta * p_r * p_r) / sig;
}

__device__ float dp_theta(float r, float theta, float E, float L) {
    float sig = sigma(r, theta);
    float dW_theta_val = dWsquare_theta(theta, E, L);
    return -dW_theta_val / (2.0f * sig);
}

// ---------------------------------------------------------------------
// RK4 integration using c_a loop to compute k coefficients
// The state vector y has 5 components.
__device__ void rk4(float* y, float h, float E, float L, float k_val) {
    float k[4][5];   // k coefficients for the 4 stages
    float y_temp[5]; // temporary storage

    // Loop over the 4 stages
#pragma unroll
    for (int stage = 0; stage < 4; ++stage) {
        float factor = (stage == 0) ? 0.0f : (stage == 3 ? 1.0f : 0.5f);
        // Compute temporary state: y_temp = y + factor * h * (previous k)
        // For stage 0 we simply have y_temp = y.
#pragma unroll
        for (int i = 0; i < 5; ++i)
            y_temp[i] = y[i] + (stage == 0 ? 0.0f : factor * h * k[stage - 1][i]);

        // Compute the derivatives at y_temp
        k[stage][0] = dr_func(y_temp[0], y_temp[1], y_temp[3]);
        k[stage][1] = dtheta_func(y_temp[0], y_temp[1], y_temp[4]);
        k[stage][2] = dphi_func(y_temp[0], y_temp[1], E, L);
        k[stage][3] = dp_r(y_temp[0], y_temp[1], y_temp[3], E, L, k_val);
        k[stage][4] = dp_theta(y_temp[0], y_temp[1], E, L);
    }
    // Combine the stages
#pragma unroll
    for (int i = 0; i < 5; ++i) {
        y[i] += h / 6.0f * (k[0][i] + 2.0f * k[1][i] + 2.0f * k[2][i] + k[3][i]);
    }
}

// ---------------------------------------------------------------------
// Kernel: each thread simulates one ray.
// Input initial conditions are in the order:
// [r0, theta0, phi0, dr0, dtheta0, dphi0]
// The output trajectory (state vector per step) and the number of steps per ray
// are stored in contiguous device memory.
__global__ void simulateRayKernel(float3 pos, size_t num_rays_per_dim, float* lookup_table) {
    //printf("%.2f %.2f %.2f \n", pos.x ,pos.y, pos.z);
    int const idx = blockIdx.x * blockDim.x + threadIdx.x;
    int const num_rays = num_rays_per_dim * num_rays_per_dim;
    if (idx >= num_rays) return;

    int const idx_theta = idx / num_rays_per_dim;
    int const idx_phi = idx % num_rays_per_dim;

    float theta = (M_PI * idx_theta) / num_rays_per_dim;
    float phi = (2.0f * M_PI * idx_phi) / num_rays_per_dim;

    // @TODO: (Investigate); Might need to rotate outgoing dirs to account for camera orientation
    float3 camPos  = make_float3(pos.x, pos.y, pos.z);  // camera world pos
    float3 forward = normalizef3(make_float3(
        -camPos.x,   // since modelCenter == (0,0,0)
        -camPos.y,
        -camPos.z
    ));

    float3 right = normalizef3(crossf3(forward, worldUp));
    float3 upVec = crossf3(right, forward);

    // now build your ray as before:
    float sinT = sinf(theta), cosT = cosf(theta);
    float sinP = sinf(phi),   cosP = cosf(phi);

    float3 dir = 
        sinT * ( cosP * right + sinP * upVec )
      + cosT * forward;

    dir = normalizef3(dir);

    float const x_vel = M_C * dir.x;
    float const y_vel = M_C * dir.y;
    float const z_vel = M_C * dir.z;

    float const A = c_a * c_rs / 2;

    float bl[6];
    cartesian_to_boyer_lindquist(pos.x, x_vel, pos.y, y_vel, pos.z, z_vel, A, bl);

    float const r0 = 2.0f / c_rs * bl[0];
    float const theta0 = bl[2];
    float const phi0 = bl[4];
    float const dr0 = bl[1] / M_C;
    float const dtheta0 = bl[3] * c_rs / (2.0f * M_C);
    float const dphi0 = bl[5] * c_rs / (2.0f * M_C);

    // Compute conserved quantities using Kerr equations.
    float E = E_func(r0, theta0, dr0, dtheta0, dphi0);
    float L = L_func(r0, theta0, dphi0, E);
    float k_val = k_func(r0, theta0, dr0, E, L);

    // Compute initial momenta.
    float S = sigma(r0, theta0);
    float p_r0 = S * dr0 / delta_r(r0);
    float p_theta0 = S * dtheta0;

    // Set up the initial state vector: [r, theta, phi, p_r, p_theta]
    float y[5];
    y[0] = r0; y[1] = theta0; y[2] = phi0; y[3] = p_r0; y[4] = p_theta0;
    //printf("%.2f, %.2f\n", theta0, phi0);

    // Pointer to this ray's lookup data. @TODO Correct index calculation old form trejectory
    float* entry = &lookup_table[idx * (1 + c_layers) * 2];

    int idx_entry = 0;
    entry[idx_entry] = theta;
    entry[idx_entry + 1] = phi;
    idx_entry += 2;
    for (int step = 0; step < c_num_steps; step++) {
        // Terminate integration if ray is inside the horizon or outside the environment.
        if (y[0] < 2.f) {
            while (idx_entry <= c_layers) {
                entry[idx_entry] = nanf("");
                entry[idx_entry + 1] = nanf("");
                idx_entry += 2;
            }
            break;
        }
        else if (y[0] > 100.f) { //TODO Check collision with the correct env map and save to entry
            entry[idx_entry] = y[1];
            entry[idx_entry + 1] = y[2];
            idx_entry += 2;
            if (idx_entry > c_layers) {
                break;
            }
        }
        // Advance one RK4 step.
        rk4(y, c_h, E, L, k_val);

    }
    if (idx_entry <= c_layers) {
        entry[idx_entry] = y[1];
        entry[idx_entry + 1] = y[2];
        idx_entry += 2;
    }
}

// ---------------------------------------------------------------------
// Exported function for DLL interface
// This function is called from Python via c_a DLL (or shared library).
// It accepts the number of rays, number of integration steps, and an array
// of initial conditions (size: num_rays * 6). It outputs the trajectory data
// (num_rays * num_steps * 5 float values) and the number of steps for each ray.
void kerr(float x, float y, float z, float rs, float Kerr, size_t num_rays_per_dim, size_t num_steps, std::vector<float>& lookup_table_host) {
    // Calculate sizes for memory allocation.

    size_t num_rays = num_rays_per_dim * num_rays_per_dim;
    size_t lookup_size = num_rays * 4 * sizeof(float);

    hipMemcpyToSymbol(HIP_SYMBOL(c_a), &Kerr, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(c_rs), &rs, sizeof(float));
    // Allocate device memory.
    float* d_lookup_table = nullptr;
    hipMalloc(&d_lookup_table, lookup_size);

    // Determine kernel launch configuration.
    int threadsPerBlock = 256;
    int blocks = (int)((num_rays + threadsPerBlock - 1) / threadsPerBlock);

    // Launch the simulation kernel.
    simulateRayKernel << <blocks, threadsPerBlock >> > (make_float3(x, y, z), num_rays_per_dim, d_lookup_table);
    hipDeviceSynchronize();

    // Copy the results back to host.
    lookup_table_host.resize(num_rays * 4);
    hipMemcpy(lookup_table_host.data(), d_lookup_table, lookup_size, hipMemcpyDeviceToHost);

    // Free device memory.
    hipFree(d_lookup_table);
}
