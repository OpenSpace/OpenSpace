#include "hip/hip_runtime.h"
#include <modules/blackhole/cuda/kerr.h>
#include <hip/hip_runtime.h>
#include <>
#include <vector_functions.h>

constexpr float PI = 3.1415926535897932384626433832795f;
constexpr float C = 299792458.0f;                            // Speed of light

#define HORIZION nanf("")
#define MAX_LAYERS 8

constexpr  float DISK = -1337.0f;
// ---------------------------------------------------------------------
constexpr float M = 1.0f;                                   // Mass parameter
constexpr float EPSILON = 1e-8;                             // Numerical tolerance

constexpr bool ACCRETION_DISK_ENABLED = true;
constexpr float ACCRETION_DISK_INNER_RADIUS = 6.0f;         // in Schwarzschild radius units
constexpr float ACCRETION_DISK_OUTER_RADIUS = 20.0f;        // in Schwarzschild radius units
constexpr float ACCRETION_DISK_TOLERANCE_THETA = 0.01f;     // small tolerance around theta = pi/2

__constant__ float c_a = 0.99f;
__constant__ unsigned int c_layers = 1;
__constant__ unsigned int c_num_steps = 5000;
__constant__ float c_env_r_values[MAX_LAYERS];
__constant__ float c_h = 0.1f;                              // Integration step size
__constant__ float c_rs = 1.0f;                              // Schwarzschild radius
__constant__ float3 c_world_up = { 0.0f, 0.0f, 1.0f };
__constant__ float3 c_forward = { 0.0f, 1.0f, 0.0f };


// ---------------------------------------------------------------------
// Coordinate convertion functions

// helper math (as before)
__device__ float3 crossf3(const float3& a, const float3& b) {
    return make_float3(
        a.y * b.z - a.z * b.y,
        a.z * b.x - a.x * b.z,
        a.x * b.y - a.y * b.x
    );
}

__device__ float3 normalizef3(const float3& v) {
    float len2 = v.x * v.x + v.y * v.y + v.z * v.z;
    float invLen = 1.0f / sqrtf(len2);
    return make_float3(v.x * invLen,
        v.y * invLen,
        v.z * invLen);
}


__device__ inline float3 operator*(const float3& v, float s) {
    return make_float3(v.x * s, v.y * s, v.z * s);
}

__device__ inline float3 operator*(float s, const float3& v) {
    return make_float3(v.x * s, v.y * s, v.z * s);
}

__device__ inline float3 operator+(const float3& a, const float3& b) {
    return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__device__ float3 spherical_to_cartesian(float r, float theta, float phi) {
    return make_float3(
        r * sinf(theta) * cosf(phi),
        r * sinf(theta) * sinf(phi),
        r * cosf(theta)
    );
}

__device__ void cartesian_to_boyer_lindquist(float x, float x_vel,
    float y, float y_vel,
    float z, float z_vel,
    float A, float* out) {
    double r2 = x * x + y * y + z * z;
    double A2 = A * A;
    double root = sqrt(A2 * (A2 - 2.0 * (x * x + y * y) + 2.0 * z * z) + r2 * r2);
    double radius = sqrt((-A2 + r2 + root) * 0.5);

    float azimuthal_angle = atan2f(y, x);
    float polar_angle = acosf(z / radius);

    double denom = 2.0 * radius * radius + A2 - r2;
    double radius_velocity = (radius * (x * x_vel + y * y_vel + z * z_vel)) / denom +
        A2 * z * z_vel / (radius * denom);

    float polar_denom = radius * sqrtf(radius * radius - z * z);
    float polar_velocity = (z * radius_velocity - z_vel * radius) / polar_denom;

    float azimuthal_velocity = (y_vel * x - x_vel * y) / (x * x + y * y);

    out[0] = radius;
    out[1] = radius_velocity;
    out[2] = polar_angle;
    out[3] = polar_velocity;
    out[4] = azimuthal_angle;
    out[5] = azimuthal_velocity;
}

__device__ inline float wrapPi(float x) {
    return x - 2.0f * PI * floorf((x + PI) / (2.0f * PI));
}

// ---------------------------------------------------------------------
// Kerr metric helper functions

__device__ float sigma(float r, float theta) {
    float cos_theta = cos(theta);
    return r * r + c_a * c_a * cos_theta * cos_theta;
}

__device__ float delta_r(float r) {
    return r * r + c_a * c_a - 2.0f * M * r;
}

__device__ float ddelta_r(float r) {
    return 2.0f * (r - M);
}

// ---------------------------------------------------------------------
// Functions W_r, W_theta and their derivatives

__device__ float W_r(float r, float E, float L) {
    return E * (r * r + c_a * c_a) - c_a * L;
}

__device__ float dWsquare_r(float r, float E, float L) {
    float W = W_r(r, E, L);
    float dW_dr = 2.0f * E * r;
    return 2.0f * W * dW_dr;
}

__device__ float W_theta(float theta, float E, float L) {
    float sin_theta = sin(theta);
    sin_theta = fmax(sin_theta, EPSILON);
    return c_a * E * sin_theta - L / sin_theta;
}

__device__ float dWsquare_theta(float theta, float E, float L) {
    float sin_theta = sin(theta);
    float cos_theta = cos(theta);
    sin_theta = fmax(sin_theta, EPSILON);
    float dW_dtheta = cos_theta * (c_a * E + L / (sin_theta * sin_theta));
    return 2.0f * W_theta(theta, E, L) * dW_dtheta;
}

// ---------------------------------------------------------------------
// Definitions of the conserved quantities and derived functions

__device__ float E_func(float r, float theta, float dr, float dtheta, float dphi) {
    float sin_theta = sin(theta);
    sin_theta = fmax(sin_theta, EPSILON);
    float delta = delta_r(r);
    float term = ((c_a * c_a * sin_theta * sin_theta - delta) * (-dr * dr / delta - dtheta * dtheta)
        + (dphi * sin_theta) * (dphi * sin_theta) * delta);
    return sqrt(term);
}

__device__ float L_func(float r, float theta, float dphi, float E) {
    float sin_theta = sin(theta);
    sin_theta = fmax(sin_theta, EPSILON);
    float delta = delta_r(r);
    float sigma_val = sigma(r, theta);
    float num = c_a * E * delta + (sigma_val * delta * dphi - c_a * E * (r * r + c_a * c_a));
    float denom = delta - c_a * c_a * sin_theta * sin_theta;
    return sin_theta * sin_theta * num / denom;
}

__device__ float k_func(float r, float theta, float dr, float E, float L) {
    float sigma_val = sigma(r, theta);
    float delta = delta_r(r);
    float W = W_r(r, E, L);
    return (W * W - sigma_val * sigma_val * dr * dr) / delta;
}

// ---------------------------------------------------------------------
// Geodesic equations: state vector y = [r, theta, phi, p_r, p_theta]
__device__ float dr_func(float r, float theta, float p_r) {
    return delta_r(r) * p_r / sigma(r, theta);
}

__device__ float dtheta_func(float r, float theta, float p_theta) {
    return p_theta / sigma(r, theta);
}

__device__ float dphi_func(float r, float theta, float E, float L) {
    float sig = sigma(r, theta);
    float delta = delta_r(r);
    float sin_theta = sin(theta);
    sin_theta = fmax(sin_theta, EPSILON);
    return (c_a * W_r(r, E, L) / delta - W_theta(theta, E, L) / sin_theta) / sig;
}

__device__ float dp_r(float r, float theta, float p_r, float E, float L, float k_val) {
    float sig = sigma(r, theta);
    float delta = delta_r(r);
    float d_delta = ddelta_r(r);
    float dW2 = dWsquare_r(r, E, L);
    float num = dW2 - d_delta * k_val;
    return (num / (2.0f * delta) - d_delta * p_r * p_r) / sig;
}

__device__ float dp_theta(float r, float theta, float E, float L) {
    float sig = sigma(r, theta);
    float dW_theta_val = dWsquare_theta(theta, E, L);
    return -dW_theta_val / (2.0f * sig);
}

// @TODO: Might need to do a line segment between points
__device__ bool check_accretion_disk_collision(float r, float theta) {
    if (ACCRETION_DISK_ENABLED && r >= ACCRETION_DISK_INNER_RADIUS && r <= ACCRETION_DISK_OUTER_RADIUS) {
        if (fabs(theta - PI / 2.0f) < ACCRETION_DISK_TOLERANCE_THETA) {
            return true;
        }
    }
    return false;
}


// ---------------------------------------------------------------------
// RK4 integration using c_a loop to compute k coefficients
// The state vector y has 5 components.
__device__ void rk4(float* y, float h, float E, float L, float k_val) {
    float k[4][5];   // k coefficients for the 4 stages
    float y_temp[5]; // temporary storage

    // Loop over the 4 stages
#pragma unroll
    for (int stage = 0; stage < 4; ++stage) {
        float factor = (stage == 0) ? 0.0f : (stage == 3 ? 1.0f : 0.5f);
        // Compute temporary state: y_temp = y + factor * h * (previous k)
        // For stage 0 we simply have y_temp = y.
#pragma unroll
        for (int i = 0; i < 5; ++i)
            y_temp[i] = y[i] + (stage == 0 ? 0.0f : factor * h * k[stage - 1][i]);

        // Compute the derivatives at y_temp
        k[stage][0] = dr_func(y_temp[0], y_temp[1], y_temp[3]);
        k[stage][1] = dtheta_func(y_temp[0], y_temp[1], y_temp[4]);
        k[stage][2] = dphi_func(y_temp[0], y_temp[1], E, L);
        k[stage][3] = dp_r(y_temp[0], y_temp[1], y_temp[3], E, L, k_val);
        k[stage][4] = dp_theta(y_temp[0], y_temp[1], E, L);
    }
    // Combine the stages
#pragma unroll
    for (int i = 0; i < 5; ++i) {
        y[i] += h / 6.0f * (k[0][i] + 2.0f * k[1][i] + 2.0f * k[2][i] + k[3][i]);
    }
}

// ---------------------------------------------------------------------
// Kernel: each thread simulates one ray.
// Input initial conditions are in the order:
// [r0, theta0, phi0, dr0, dtheta0, dphi0]
// The output trajectory (state vector per step) and the number of steps per ray
// are stored in contiguous device memory.
__global__ void simulateRayKernel(float3 pos, size_t num_rays_per_dim, float* lookup_table) {
    //printf("%.2f %.2f %.2f \n", pos.x ,pos.y, pos.z);
    int const idx = blockIdx.x * blockDim.x + threadIdx.x;
    int const num_rays = num_rays_per_dim * num_rays_per_dim;
    if (idx >= num_rays) return;

    int const idx_theta = idx / num_rays_per_dim;
    int const idx_phi = idx % num_rays_per_dim;

    float theta = PI - (PI * idx_theta) / num_rays_per_dim;
    float phi = (2.0f * PI * idx_phi) / num_rays_per_dim - PI;

    // @TODO: (Investigate); Might need to rotate outgoing dirs to account for camera orientation
    float3 camPos = make_float3(pos.x, pos.y, pos.z);  // camera world pos

    float3 right = normalizef3(crossf3(c_forward, c_world_up));
    float3 upVec = crossf3(right, c_forward);

    // now build your ray as before:
    float sinT = sinf(theta), cosT = cosf(theta);
    float sinP = sinf(phi), cosP = cosf(phi);

    float3 dir =
        sinT * (cosP * right + sinP * upVec)
        + cosT * c_forward;

    dir = normalizef3(dir);

    float const x_vel = C * dir.x;
    float const y_vel = C * dir.y;
    float const z_vel = C * dir.z;

    float const A = c_a * c_rs / 2;

    float bl[6];
    cartesian_to_boyer_lindquist(pos.x, x_vel, pos.y, y_vel, pos.z, z_vel, A, bl);

    float const r0 = 2.0f / c_rs * bl[0];
    float const theta0 = bl[2];
    float const phi0 = bl[4];
    float const dr0 = bl[1] / C;
    float const dtheta0 = bl[3] * c_rs / (2.0f * C);
    float const dphi0 = bl[5] * c_rs / (2.0f * C);

    // Compute conserved quantities using Kerr equations.
    float E = E_func(r0, theta0, dr0, dtheta0, dphi0);
    float L = L_func(r0, theta0, dphi0, E);
    float k_val = k_func(r0, theta0, dr0, E, L);

    // Compute initial momenta.
    float S = sigma(r0, theta0);
    float p_r0 = S * dr0 / delta_r(r0);
    float p_theta0 = S * dtheta0;

    // Set up the initial state vector: [r, theta, phi, p_r, p_theta]
    float y[5];
    y[0] = r0; y[1] = theta0; y[2] = phi0; y[3] = p_r0; y[4] = p_theta0;
    //printf("%.2f, %.2f\n", theta0, phi0);

    // Pointer to this ray's lookup data. @TODO Correct index calculation old form trejectory
    float* entry = &lookup_table[idx * (1 + c_layers) * 2];

    int idx_entry = 0;
    entry[idx_entry] = theta;
    entry[idx_entry + 1] = phi;
    idx_entry += 2;
    for (size_t l = 0; l < c_layers + 1; l++) {
        for (int step = 0; step < c_num_steps; step++) {
            // Terminate integration if ray is inside the horizon or outside the environment.
            if (y[0] < 2.f) {
                while (idx_entry < (c_layers + 1) * 2) {
                    entry[idx_entry] = HORIZION;
                    entry[idx_entry + 1] = HORIZION;
                    idx_entry += 2;
                }
                break;
            }
            else if (y[0] > c_env_r_values[l]) {
                entry[idx_entry] = y[1];
                entry[idx_entry + 1] = y[2];
                idx_entry += 2;
                if (idx_entry > (c_layers + 1) * 2) {
                    break;
                }
            }
            else if (check_accretion_disk_collision(y[0], y[1])) {
                while (idx_entry < (c_layers + 1) * 2) {
                    entry[idx_entry] = DISK;
                    entry[idx_entry + 1] = 1 - abs((y[0] - ACCRETION_DISK_INNER_RADIUS) / ACCRETION_DISK_OUTER_RADIUS);
                    idx_entry += 2;
                }
                break;
            }

            // Advance one RK4 step.
            rk4(y, c_h, E, L, k_val);

            y[1] = y[1];
            y[2] = wrapPi(y[2]);
        }
    }
    if (idx_entry < (c_layers + 1) * 2) {
        entry[idx_entry] = y[1];
        entry[idx_entry + 1] = y[2];
        idx_entry += 2;
    }
}

// ---------------------------------------------------------------------
// Exported function for DLL interface
// This function is called from Python via c_a DLL (or shared library).
// It accepts the number of rays, number of integration steps, and an array
// of initial conditions (size: num_rays * 6). It outputs the trajectory data
// (num_rays * num_steps * 5 float values) and the number of steps for each ray.
void traceKerr(glm::vec3 position, float rs, float kerr, std::vector<float> env_r_values, size_t num_rays_per_dim, size_t num_steps, std::vector<float>& lookup_table_host) {
    // Calculate sizes for memory allocation.
    unsigned int const layers = env_r_values.size();
    size_t output_size = (layers + 1) * 2;
    size_t num_rays = num_rays_per_dim * num_rays_per_dim;
    size_t lookup_size = num_rays * output_size * sizeof(float);

    hipMemcpyToSymbol(HIP_SYMBOL(c_layers), &layers, sizeof(unsigned int));
    hipMemcpyToSymbol(HIP_SYMBOL(c_a), &kerr, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(c_rs), &rs, sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(
        c_env_r_values),
        env_r_values.data(),
        env_r_values.size() * sizeof(float)
    );

    // Allocate device memory.
    float* __restrict__ d_lookup_table;
    hipMalloc(&d_lookup_table, lookup_size);

    // Determine kernel launch configuration.
    int threadsPerBlock = 256;
    int blocks = (int)((num_rays + threadsPerBlock - 1) / threadsPerBlock);

    // Launch the simulation kernel.
    simulateRayKernel << <blocks, threadsPerBlock >> > (make_float3(position.x, -position.z, position.y), num_rays_per_dim, d_lookup_table);
    hipDeviceSynchronize();

    // Copy the results back to host.
    lookup_table_host.resize(num_rays * output_size);
    hipMemcpy(lookup_table_host.data(), d_lookup_table, lookup_size, hipMemcpyDeviceToHost);

    // Free device memory.
    hipFree(d_lookup_table);
}
